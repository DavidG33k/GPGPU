/**
GPGPU assignment 1: Matrix Addition in CUDA - Standard Memory/Grid Stride Loop Kernels version
    @file sm_grid_stride_loop_kernels.cu
    @author Canonaco Martina @author Gena Davide @author Morello Michele @author Oliviero Tiziana
    @version 13 October 2021 
*
Let A and B be the matrices of double-precision floating-point numbers to be added,
and C the resulting matrix; Let m = 2^12 and n=2^15 be their number of rows and columns, respectively.
*
Implement four versions of the matrix addition application in CUDA using:
    - Standard-Memory/Grid-Stride-Loop-Kernels.
*/


#include <hip/hip_runtime.h>
#include <algorithm>
#include <iostream>
#include <math.h>
using namespace std;

#define M (1 << 12) //m=2^12 = 4096
#define N (1 << 15) //n=2^15 = 32768

__global__
void matrixInit(double* A, double value)
{
  int index_x = blockIdx.x * blockDim.x + threadIdx.x;
  int index_y = blockIdx.y * blockDim.y + threadIdx.y;
  int stride_x = blockDim.x * gridDim.x;
  int stride_y = blockDim.y * gridDim.y;

  for (int i = index_x; i < M; i += stride_x)
    for (int j = index_y; j < N; j += stride_y)
        A[j*M+i]=value;
}

__global__
void matrixAdd(double *A, double *B, double *C)
{
  int index_x = blockIdx.x * blockDim.x + threadIdx.x;
  int index_y = blockIdx.y * blockDim.y + threadIdx.y;
  int stride_x = blockDim.x * gridDim.x;
  int stride_y = blockDim.y * gridDim.y;

  for (int i = index_x; i < M; i += stride_x)
    for (int j = index_y; j < N; j += stride_y)
        C[j*M+i] = A[j*M+i] + B[j*M+i];
}

void printMatrix(double* A)
{
    for(int i=0; i<M*N; i++)
        cout<<" "<<A[i]<<" ";
}

int main()
{
#pragma region //variables declaration
    double size = M * N * sizeof(double);
    cout<<"size: "<<size<<endl;
    
    dim3 dimBlock(32,32);
    dim3 dimGrid(((N+dimBlock.x-1)/dimBlock.x),((M+dimBlock.y-1)/dimBlock.y));
#pragma endregion

#pragma region //create and allocate matrix A, B and C
    //allocate dynamic matrix
    double *A, *B, *C; //host matrix

    //in standard memory we have to allocate CPU
    A = (double*)malloc(size);
    B = (double*)malloc(size);
    C = (double*)malloc(size);

    double *dev_A, *dev_B, *dev_C; //device matrix

    hipMalloc((void**)&dev_A, size);
    hipMalloc((void**)&dev_B, size);
    hipMalloc((void**)&dev_C, size);

    hipMemcpy(dev_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(dev_B, B, size, hipMemcpyHostToDevice);
#pragma endregion

#pragma region //init all the matrix with a passed value
    matrixInit<<<dimGrid, dimBlock>>>(dev_A,1.0);
    matrixInit<<<dimGrid, dimBlock>>>(dev_B,2.0);
    matrixInit<<<dimGrid, dimBlock>>>(dev_C,0.0);
#pragma endregion

#pragma region //addiction operation and print results
    matrixAdd<<<dimGrid, dimBlock>>>(dev_A, dev_B, dev_C);

    hipDeviceSynchronize();

    hipMemcpy(C, dev_C, size, hipMemcpyDeviceToHost);

    //printing resulting matrix C
    cout<<endl<<"MatrixC final"<<endl;
    //printMatrix(C);
#pragma endregion

#pragma region //check for errors (all values should be 3.0f)
    float maxError = 0;
    for (int i = 0; i < M * N; i++)
	    maxError=fmax(maxError, fabs(C[i]-3.0f));
    cout << "Max error: " << maxError << endl;
#pragma endregion

#pragma region //free cuda memory
    hipFree(dev_A); 
    hipFree(dev_B); 
    hipFree(dev_C);
#pragma region

    return 0;
}
/**
GPGPU assignment 1: Matrix Addition in CUDA - Standard Memory/Grid Stride Loop Kernels version
    @file sm_grid_stride_loop_kernels.cu
    @author Canonaco Martina @author Gena Davide @author Morello Michele
    @version 13 October 2021 
*
Let A and B be the matrices of double-precision floating-point numbers to be added,
and C the resulting matrix; Let m = 2^12 and n=2^16 be their number of rows and columns, respectively.
*
Implement four versions of the matrix addition application in CUDA using:
    - Standard-Memory/Monolithic-Kernels;
    - Standard-Memory/Grid-Stride-Loop-Kernels;
    - Unified-Memory/Monolithic-Kernels;
    - Unified-Memory/Grid-Stride-Loop-Kernels.
*/


#include <hip/hip_runtime.h>
#include <algorithm>
#include <iostream>
using namespace std;

#define M 5
#define N 5

void matrixInit(double A[][N], double value)
{
    for(int i=0; i<M; i++)
        for(int j=0; j<N; j++)
            A[i][j] = value;
}

__global__
void matrixAdd(double *A, double *B, double *C)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;

  for (int i = index; i < M*N; i += stride)
    C[i] = A[i] + B[i];
}

void printMatrix(double A[][N])
{
    for(int i=0; i<M; i++)
    {
        for(int j=0; j<N; j++)
            cout<<" "<<A[i][j]<<" ";
        cout<<endl;
    }
}

int main()
{
//variables declaration
    double size = M * N * sizeof(double);
    
    int blockSize = 256;
    int numBlocks = (M * N + blockSize - 1) / blockSize;

//create and allocate matrix A, B and C
    double A[M][N];
    double B[M][N];
    double C[M][N];

    double *dev_A, *dev_B, *dev_C;

    hipMalloc((void**)&dev_A, size);
    hipMalloc((void**)&dev_B, size);
    hipMalloc((void**)&dev_C, size);

//init all the matrix with a passed value
    matrixInit(A,1.0f);
    matrixInit(B,2.0f);
    matrixInit(C,0.0f);

    hipMemcpy(dev_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(dev_B, B, size, hipMemcpyHostToDevice);

//addiction operation and print results
    matrixAdd<<<numBlocks, blockSize>>>(dev_A, dev_B, dev_C);
    hipDeviceSynchronize();

    hipMemcpy(C, dev_C, size, hipMemcpyDeviceToHost);

//printing resulting matrix C
    cout<<endl<<"PRINT C final"<<endl;
    printMatrix(C);

//free cuda memory
    hipFree(dev_A); 
    hipFree(dev_B); 
    hipFree(dev_C);

    return 0;
}
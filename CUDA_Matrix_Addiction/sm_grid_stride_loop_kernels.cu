/**
GPGPU assignment 1: Matrix Addition in CUDA - Standard Memory/Grid Stride Loop Kernels version
    @file sm_grid_stride_loop_kernels.cu
    @author Canonaco Martina @author Gena Davide @author Morello Michele
    @version 13 October 2021 
*
Let A and B be the matrices of double-precision floating-point numbers to be added,
and C the resulting matrix; Let m = 2^12 and n=2^16 be their number of rows and columns, respectively.
*
Implement four versions of the matrix addition application in CUDA using:
    - Standard-Memory/Monolithic-Kernels;
    - Standard-Memory/Grid-Stride-Loop-Kernels;
    - Unified-Memory/Monolithic-Kernels;
    - Unified-Memory/Grid-Stride-Loop-Kernels.
*/


#include <hip/hip_runtime.h>
#include <algorithm>
#include <iostream>
using namespace std;

#define M 1240
#define N 1500

__global__
void matrixInit(double* A, double value)
{
    for(int i=0; i<M*N; i++)
        A[i] = value;
}

__global__
void matrixAdd(double *A, double *B, double *C)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;

  for (int i = index; i < M*N; i += stride)
    C[i] = A[i] + B[i];
}

void printMatrix(double A[][N])
{
    for(int i=0; i<M; i++)
    {
        for(int j=0; j<N; j++)
            cout<<" "<<A[i][j]<<" ";
        cout<<endl;
    }
}

int main()
{
//variables declaration
    double size = M * N * sizeof(double);
    
    int blockSize = 256;
    int numBlocks = (M * N + blockSize - 1) / blockSize;

//create and allocate matrix A, B and C
    //allocate dynamic matrix
    double *A, *B, *C; //host matrix

    //in standard memory we have to allocate CPU
    A = (double*)malloc(size);
    B = (double*)malloc(size);
    C = (double*)malloc(size);

    double *dev_A, *dev_B, *dev_C; //device matrix

    hipMalloc((void**)&dev_A, size);
    hipMalloc((void**)&dev_B, size);
    hipMalloc((void**)&dev_C, size);

    hipMemcpy(dev_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(dev_B, B, size, hipMemcpyHostToDevice);
    hipMemcpy(dev_C, C, size, hipMemcpyHostToDevice);

//init all the matrix with a passed value
    matrixInit<<<numBlocks, blockSize>>>(dev_A,1.0);
    matrixInit<<<numBlocks, blockSize>>>(dev_B,2.0);
    matrixInit<<<numBlocks, blockSize>>>(dev_C,0.0);
    cout<<endl<<"M-init done"<<endl;

//addiction operation and print results
    matrixAdd<<<numBlocks, blockSize>>>(dev_A, dev_B, dev_C);

cout<<endl<<"Sync starts"<<endl;
    //cudaDeviceSynchronize();
cout<<endl<<"Sync ends"<<endl;

cout<<endl<<"Copy starts"<<endl;
    hipMemcpy(C, dev_C, size, hipMemcpyDeviceToHost);
cout<<endl<<"Copy ends"<<endl;

//printing resulting matrix C
    cout<<endl<<"PRINT C final"<<endl;
    //printMatrix(C);

//free cuda memory
    hipFree(dev_A); 
    hipFree(dev_B); 
    hipFree(dev_C);

    return 0;
}
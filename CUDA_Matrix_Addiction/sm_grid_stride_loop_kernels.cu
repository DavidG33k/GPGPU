/**
GPGPU assignment 1: Matrix Addition in CUDA - Standard Memory/Grid Stride Loop Kernels version
    @file sm_grid_stride_loop_kernels.cu
    @author Canonaco Martina @author Gena Davide @author Morello Michele @author Oliviero Tiziana
    @version 13 October 2021 
*
Let A and B be the matrices of double-precision floating-point numbers to be added,
and C the resulting matrix; Let m = 2^12 and n=2^15 be their number of rows and columns, respectively.
*
Implement four versions of the matrix addition application in CUDA using:
    - Standard-Memory/Grid-Stride-Loop-Kernels.
*/


#include <hip/hip_runtime.h>
#include <algorithm>
#include <iostream>
using namespace std;

#define M 4096 //m=2^12 = 4096
#define N 32768 //n=2^15 = 32768

__global__
void matrixInit(double* A, double value)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;

  for (int i = index; i < M*N; i += stride)
    A[i]=value;
}

__global__
void matrixAdd(double *A, double *B, double *C)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;
  int stride = blockDim.x * gridDim.x;

  for (int i = index; i < M*N; i += stride)
    C[i] = A[i] + B[i];
}

void printMatrix(double* A)
{
    for(int i=0; i<M*N; i++)
        cout<<" "<<A[i]<<" ";
}

int main()
{
//variables declaration
    double size = M * N * sizeof(double);
    cout<<"size: "<<size<<endl;
    
    int blockSize = 256;
    int numBlocks = (M * N + blockSize - 1) / blockSize;

//create and allocate matrix A, B and C
    //allocate dynamic matrix
    double *A, *B, *C; //host matrix

    //in standard memory we have to allocate CPU
    A = (double*)malloc(size);
    B = (double*)malloc(size);
    C = (double*)malloc(size);

    double *dev_A, *dev_B, *dev_C; //device matrix

    hipMalloc((void**)&dev_A, size);
    hipMalloc((void**)&dev_B, size);
    hipMalloc((void**)&dev_C, size);

    hipMemcpy(dev_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(dev_B, B, size, hipMemcpyHostToDevice);

//init all the matrix with a passed value
    matrixInit<<<numBlocks, blockSize>>>(dev_A,1.0);
    matrixInit<<<numBlocks, blockSize>>>(dev_B,2.0);
    matrixInit<<<numBlocks, blockSize>>>(dev_C,0.0);
    cout<<endl<<"M-init done"<<endl;

//addiction operation and print results
    cout<<endl<<"add starts"<<endl;
    matrixAdd<<<numBlocks, blockSize>>>(dev_A, dev_B, dev_C);
    cout<<endl<<"add ends"<<endl;

    // cout<<endl<<"synch starts"<<endl;
    // cudaDeviceSynchronize();
    // cout<<endl<<"synch ends"<<endl;

    cout<<endl<<"final copy starts"<<endl;
    hipMemcpy(C, dev_C, size, hipMemcpyDeviceToHost);
    cout<<endl<<"final copy ends"<<endl;

//printing resulting matrix C
    cout<<endl<<"PRINT C final"<<endl;
    //printMatrix(C);

//free cuda memory
    hipFree(dev_A); 
    hipFree(dev_B); 
    hipFree(dev_C);

    return 0;
}
/**
GPGPU assignment 1: Matrix Addition in CUDA - Unified Memory/Monolithic Kernels version
    @file um_monolithic_kernels.cu
    @author Canonaco Martina @author Gena Davide @author Morello Michele @author Oliviero Tiziana
    @version 13 October 2021 
*
Let A and B be the matrices of double-precision floating-point numbers to be added,
and C the resulting matrix; Let m = 2^12 and n=2^15 be their number of rows and columns, respectively.
*
Implement four versions of the matrix addition application in CUDA using:
    - Unified-Memory/Monolithic-Kernels.
*/


#include <hip/hip_runtime.h>
#include<iostream>
#include<math.h>
using namespace std;

#define M 4096 //m=2^12 = 4096
#define N 32768 //n=2^15 = 32768

__global__
void matrixInit(double* A, double value)
{
    for(int i=0; i<M*N; i++)
        A[i] = value;
}

__global__
void matrixAdd(double* A, double* B, double* C)
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    int index = col + row * N;

    if (col < N && row < M) {
        C[index] = A[index] + B[index];
    }
}

void printMatrix(double* A)
{
    for(int i=0; i<M*N; i++)
        cout<<" "<<A[i]<<" ";
}

int main()
{
//variables declaration
    double size = M * N * sizeof(double); //expect a size in bytes
    cout<<"size: "<<size<<endl;

    dim3 dimBlock(16,16);
    dim3 dimGrid(((N+dimBlock.x-1)/dimBlock.x),((M+dimBlock.y-1)/dimBlock.y));

//create and allocate matrix A, B and C
    double* A; hipMallocManaged(&A, size);
    double* B; hipMallocManaged(&B, size);
    double* C; hipMallocManaged(&C, size);

//init all the matrix with a passed value
    matrixInit<<<dimGrid, dimBlock>>>(A,1.0);
    matrixInit<<<dimGrid, dimBlock>>>(B,2.0);
    matrixInit<<<dimGrid, dimBlock>>>(C,0.0);
    cout<<endl<<"M-init done"<<endl;
 
 //addiction operation and print results
    cout<<endl<<"add starts"<<endl;
    matrixAdd<<<dimGrid, dimBlock>>>(A, B, C);

    cout<<endl<<"Sync starts"<<endl;
    hipDeviceSynchronize();
    cout<<endl<<"Sync ends"<<endl;

//printing resulting matrix C
    cout<<endl<<"PRINT C final"<<endl;
    //printMatrix(values_C);

//free cuda memory
    hipFree(A); 
    hipFree(B); 
    hipFree(C);
    
    return 0;
}
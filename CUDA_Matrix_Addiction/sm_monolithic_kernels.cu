/**
GPGPU assignment 1: Matrix Addition in CUDA - Standard Memory/Monolithic Kernels version
    @file sm_monolithic_kernels.cu
    @author Canonaco Martina @author Gena Davide @author Morello Michele @author Oliviero Tiziana
    @version 13 October 2021 
*
Let A and B be the matrices of double-precision floating-point numbers to be added,
and C the resulting matrix; Let m = 2^12 and n=2^15 be their number of rows and columns, respectively.
*
Implement four versions of the matrix addition application in CUDA using:
    - Standard-Memory/Monolithic-Kernels.
*/


#include <hip/hip_runtime.h>
#include<iostream>
#include<math.h>
using namespace std;

#define M (1 << 12) //m=2^12 = 4096
#define N (1 << 15) //n=2^15 = 32768

__global__
void matrixInit(double* A, double value)
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    int index = col + row * N;

    if (col < N && row < M) {
        A[index]=value;
    }
}

__global__
void matrixAdd(double* A, double* B, double* C)
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    int index = col + row * N;

    if (col < N && row < M) {
        C[index] = A[index] + B[index];
    }
}

void printMatrix(double* A)
{
    for(int i=0; i<M*N; i++)
        cout<<" "<<A[i]<<" ";
}

int main()
{
//variables declaration
    double size = M * N * sizeof(double); //expect a size in bytes
    cout<<"size: "<<size<<endl;

    dim3 dimBlock(32,32);
    dim3 dimGrid(((N+dimBlock.x-1)/dimBlock.x),((M+dimBlock.y-1)/dimBlock.y));

//create and allocate matrix A, B and C
    //allocate dynamic matrix
    double *A, *B, *C; //host matrix

    //in standard memory we have to allocate CPU
    A = (double*)malloc(size);
    B = (double*)malloc(size);
    C = (double*)malloc(size);

    double *dev_A, *dev_B, *dev_C; //device matrix

    hipMalloc((void**)&dev_A, size);
    hipMalloc((void**)&dev_B, size);
    hipMalloc((void**)&dev_C, size);

    hipMemcpy(dev_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(dev_B, B, size, hipMemcpyHostToDevice);

//init all the matrix with a passed value
    matrixInit<<<dimGrid, dimBlock>>>(dev_A,1.0);
    matrixInit<<<dimGrid, dimBlock>>>(dev_B,2.0);
    matrixInit<<<dimGrid, dimBlock>>>(dev_C,0.0);
    cout<<endl<<"M-init done"<<endl;
 
//addiction operation and print results
    cout<<endl<<"add starts"<<endl;
    matrixAdd<<<dimGrid, dimBlock>>>(dev_A, dev_B, dev_C);
    cout<<endl<<"add ends"<<endl;

    // cout<<endl<<"synch starts"<<endl;
    // cudaDeviceSynchronize();
    // cout<<endl<<"synch ends"<<endl;

    cout<<endl<<"final copy starts"<<endl;
    hipMemcpy(C, dev_C, size, hipMemcpyDeviceToHost);
    cout<<endl<<"final copy ends"<<endl;
//printing resulting matrix C
    cout<<endl<<"PRINT C final"<<endl;
    //printMatrix(C);

// Check for errors (all values should be 3.0f)	
    float maxError = 0;
    for (int i = 0; i < M * N; i++)
	maxError=fmax(maxError, fabs(C[i]-3.0f));
    cout << endl << "Max error: " << maxError << endl;

//free cuda memory
    hipFree(dev_A); 
    hipFree(dev_B); 
    hipFree(dev_C);
    
    return 0;
}
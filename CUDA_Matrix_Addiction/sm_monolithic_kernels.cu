/**
GPGPU assignment 1: Matrix Addition in CUDA - Standard Memory/Monolithic Kernels version
    @file sm_monolithic_kernels.cu
    @author Canonaco Martina @author Gena Davide @author Morello Michele @author Oliviero Tiziana
    @version 13 October 2021 
*
Let A and B be the matrices of double-precision floating-point numbers to be added,
and C the resulting matrix; Let m = 2^12 and n=2^16 be their number of rows and columns, respectively.
*
Implement four versions of the matrix addition application in CUDA using:
    - Standard-Memory/Monolithic-Kernels;
    - Standard-Memory/Grid-Stride-Loop-Kernels;
    - Unified-Memory/Monolithic-Kernels;
    - Unified-Memory/Grid-Stride-Loop-Kernels.
*/


#include <hip/hip_runtime.h>
#include<iostream>
#include<math.h>
using namespace std;

#define M 1000
#define N 900

__global__
void matrixInit(double* A, double value)
{
    for(int i=0; i<M*N; i++)
        A[i] = value;
}

__global__
void matrixAdd(double* A, double* B, double* C)
{
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int row = blockIdx.y * blockDim.y + threadIdx.y;

    int index = col + row * N;

    if (col < N && row < M) {
        C[index] = A[index] + B[index];
    }
}

void printMatrix(double A[][N])
{
    for(int i=0; i<M; i++)
    {
        for(int j=0; j<N; j++)
            cout<<" "<<A[i][j]<<" ";
        cout<<endl;
    }
}

int main()
{
//variables declaration
    int size = M * N * sizeof(double); //expect a size in bytes

    dim3 dimBlock(16,16);
    dim3 dimGrid(((N+dimBlock.x-1)/dimBlock.x),((M+dimBlock.y-1)/dimBlock.y));

//create and allocate matrix A, B and C
    //allocate dynamic matrix
    double *A, *B, *C; //host matrix

    //in standard memory we have to allocate CPU
    A = (double*)malloc(size);
    B = (double*)malloc(size);
    C = (double*)malloc(size);

    double *dev_A, *dev_B, *dev_C; //device matrix

    hipMalloc((void**)&dev_A, size);
    hipMalloc((void**)&dev_B, size);
    hipMalloc((void**)&dev_C, size);

    hipMemcpy(dev_A, A, size, hipMemcpyHostToDevice);
    hipMemcpy(dev_B, B, size, hipMemcpyHostToDevice);
    hipMemcpy(dev_C, C, size, hipMemcpyHostToDevice);

//init all the matrix with a passed value
    matrixInit<<<dimGrid, dimBlock>>>(dev_A,1.0);
    matrixInit<<<dimGrid, dimBlock>>>(dev_B,2.0);
    matrixInit<<<dimGrid, dimBlock>>>(dev_C,0.0);
    cout<<endl<<"M-init done"<<endl;
 
//addiction operation and print results
cout<<endl<<"Addiction starts"<<endl;
    matrixAdd<<<dimGrid, dimBlock>>>(dev_A, dev_B, dev_C);
cout<<endl<<"Addiction ends"<<endl;
    hipDeviceSynchronize();

    hipMemcpy(C, dev_C, size, hipMemcpyDeviceToHost);

//printing resulting matrix C
    cout<<endl<<"PRINT C final"<<endl;
    //printMatrix(C);

//free cuda memory
    hipFree(dev_A); 
    hipFree(dev_B); 
    hipFree(dev_C);
    
    return 0;
}
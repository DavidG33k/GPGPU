#include "hip/hip_runtime.h"
/**
GPGPU assignment 2: Matrix Multiplication in CUDA
    @file matrix_multiplication_serial.cpp
    @author Canonaco Martina @author Gena Davide @author Morello Michele @author Oliviero Tiziana
    @version 03 November 2021 
*A serial implementation of the matrix multiplication algorithm in C/C++.
 - dims of M = 2000x500
 - dims of N = 500x2000
*/

#include <algorithm>
#include <iostream>
#include <math.h>
#include <stdlib.h>
using namespace std;

#define d1 2000
#define d2 500
#define d3 2000

__global__
void matrixInit(float* A, float value, int raw, int col)
{
  int index_x = blockIdx.x * blockDim.x + threadIdx.x;
  int index_y = blockIdx.y * blockDim.y + threadIdx.y;
  int stride_x = blockDim.x * gridDim.x;
  int stride_y = blockDim.y * gridDim.y;

  for (int i = index_x; i < raw; i += stride_x)
    for (int j = index_y; j < col; j += stride_y)
        A[j*raw+i]=value;
}

__global__
void matrixMulti(float* dev_M, float* dev_N, float* dev_P, unsigned j, unsigned k, unsigned l, int tileSize)
{
     
     __shared__
     float Mds[tileSize][tileSize];
     __shared__
     float Nds[tileSize][tileSize];
     
     
     int tc = threadIdx.x;
     int tr = threadIdx.y;
     int Row = blockIdx.y * tileSize + tr;
     int Col = blockIdx.x * tileSize + tc;

     float Pvalue = 0.0f;
     for(int ph = 0; ph < k/tileSize; ++ph)
     {
          if((Row < j) && (ph * tileSize + tc) < k)
               Mds[tr][tc] = dev_M[Row * k + ph * tileSize +tc];
          else
               Mds[tr][tc] = 0;
          if((ph * tileSize +tr) < k && Col < l)
               Nds[tr][tc] = dev_N[ph * tileSize + tc];
          else
               Nds[tr][tc] = 0;
          __syncthreads();
          for(int i = 0; i < tileSize; ++i)
               Pvalue += Mds[tr][i] * Nds[i][tc];
          __syncthreads();
     }
     if((Row < j) && (Col < l))
          dev_P[Row * l + Col] = Pvalue;
}

int main(int argc, char* argv[])
{
#pragma region //managing argv && argc
	int blockSize,
	int tileSize;
    if(argc != 2){
    	cout<<"no Block Size declared!"<<endl;
    	return 0;
    }
    
    blockSize = atoi(argv[1]);
    
    if(blockSize!=8 && blockSize!=16 && blockSize!=32){
    	cout<<"Invalid Block Size!"<<endl;
    	return 0;
    }
    
    tileSize = blockSize;
    cout<<"TILE SIZE= "<<tileSize<<endl;

#pragma region //variables declaration
    float size_M = d1 * d2 * sizeof(float);
    float size_N = d2 * d1 * sizeof(float);
    float size_P = d1 * d3 * sizeof(float);
    cout<<"size of M: "<<size_M<<endl;
    cout<<"size of N: "<<size_N<<endl;
    cout<<"size of P: "<<size_P<<endl;
    
    
    dim3 dimBlock(blockSize,blockSize);
        
    dim3 dimGrid(((d1+dimBlock.x-1)/dimBlock.x),((d3+dimBlock.y-1)/dimBlock.y));
#pragma endregion

#pragma region //create and allocate matrix A, B and C
    float* M; hipMallocManaged(&M, size_M);
    float* N; hipMallocManaged(&N, size_N);
    float* P; hipMallocManaged(&P, size_P);
#pragma endregion

#pragma region //init all the matrix with a passed value
   
    
    matrixInit<<<dimGrid, dimBlock>>>(M,2.0f, d1, d2);
    matrixInit<<<dimGrid, dimBlock>>>(N,3.0f, d2, d1);
    matrixInit<<<dimGrid, dimBlock>>>(P,0.0f, d1, d3);
#pragma endregion

#pragma region //multiplication operation
    matrixMulti<<<dimGrid, dimBlock>>>(M, N, P, d1, d2, d3, tileSize);

    hipDeviceSynchronize();
#pragma endregion


cout<<"P[0] = "<<P[0]<<endl;

#pragma region //check for errors (all values should be 3000.0f)
    float maxError = 0;
    for (int i = 0; i < d1 * d3; i++)
	    maxError=fmax(maxError, fabs(P[i]-3000.0f));
    cout << "Max error: " << maxError << endl;
#pragma endregion

#pragma region //free cuda memory
    hipFree(M); 
    hipFree(N); 
    hipFree(P);
#pragma region

    return 0;
}

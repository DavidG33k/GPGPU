#include "hip/hip_runtime.h"
/**
GPGPU assignment 3: Parallel image convolutional filtering
    @file global_memory_implementation.cu
    @author Canonaco Martina @author Gena Davide @author Morello Michele @author Oliviero Tiziana
    @version 02 December 2021 
*/

#include"../lib/lodepng.h"
#include<iostream>
#include<time.h>
#include<assert.h>
#include<vector>
using namespace std;

#define WIDTH 800
#define HEIGHT 800
#define BLUR_SIZE 20
#define NUM_CHANNELS 3
#define R 0
#define G 1
#define B 2

#define input_path "../images/myImage.png"
#define output_path "../images/encoded_image.png"

__global__ void blurKernel(unsigned char* dev_in, unsigned char* dev_out, int w, int h, int num_channels, int channel)
{
    int Col = blockIdx.x * blockDim.x + threadIdx.x;
    int Row = blockIdx.y * blockDim.y + threadIdx.y;

    if(Col < w && Row < h) {
        int pixVal = 0;
        int pixels = 0;

        for(int blurRow = -BLUR_SIZE; blurRow < BLUR_SIZE+1; ++blurRow) {
            for(int blurCol = -BLUR_SIZE; blurCol < BLUR_SIZE+1; ++blurCol) {
                int curRow = Row + blurRow;
                int curCol = Col + blurCol;

                if(curRow > -1 && curRow < h && curCol > -1 && curCol < w) {
                    pixVal += dev_in[curRow * w * num_channels + curCol * num_channels + channel];
                    pixels++;
                }
            }
        }

        dev_out[Row * w * num_channels + Col * num_channels + channel] = (unsigned char)(pixVal / pixels);
    }
}


int main(int argc, char* argv[])
{
   
#pragma region  //managing argv, argc, time
    clock_t start, end;
    int blockSize;
    assert(argc == 2);
    blockSize = atoi(argv[1]);
    assert(blockSize==8 || blockSize==16 || blockSize==32);    
#pragma endregion

#pragma region //variables declaration
    start=clock();
    dim3 dimBlock(blockSize,blockSize);
    dim3 dimGrid(ceil(WIDTH/dimBlock.x), ceil(HEIGHT/dimBlock.y));
#pragma endregion

#pragma region 
   
    vector<unsigned char> imageInput;
    vector<unsigned char> imageOutput;

    unsigned int w = WIDTH;
    unsigned int h = HEIGHT;

    lodepng::decode(imageInput, w, h, input_path, LCT_RGB);
    cout << "buffer input size: " << imageInput.size() << endl;
    
    int size = imageInput.size() * sizeof(unsigned char);

    unsigned char *in, *out;
    in = (unsigned char*)malloc(size);
    out = (unsigned char*)malloc(size);

    for(int i=0; i<imageInput.size(); i++)
        in[i] = imageInput[i];

    cout << "ci sono" << endl;

    unsigned char* Dev_Input_Image = NULL;
    unsigned char* Dev_Output_Image = NULL;
    hipMalloc((void**)&Dev_Input_Image, size);
    hipMalloc((void**)&Dev_Output_Image, size);

    hipMemcpy(Dev_Input_Image, in, size, hipMemcpyHostToDevice);

    cout << "ci sono" << endl;

    blurKernel<<<dimGrid, dimBlock>>>(Dev_Input_Image, Dev_Output_Image, w, h, NUM_CHANNELS, R);
    blurKernel<<<dimGrid, dimBlock>>>(Dev_Input_Image, Dev_Output_Image, w, h, NUM_CHANNELS, G);
    blurKernel<<<dimGrid, dimBlock>>>(Dev_Input_Image, Dev_Output_Image, w, h, NUM_CHANNELS, B);

    hipDeviceSynchronize();

    cout << "ci sono" << endl;

    hipMemcpy(out, Dev_Output_Image, size, hipMemcpyDeviceToHost);

    for(int i=0; i<imageInput.size(); i++)
        imageOutput.push_back(out[i]);

     cout << "buffer output size: " << imageOutput.size() << endl;

    cout << "ci sono" << endl;

    hipFree(Dev_Input_Image);
    hipFree(Dev_Output_Image);

    lodepng::encode(output_path, imageOutput, w, h, LCT_RGB);

#pragma endregion


#pragma region  //checking errors

#pragma endregion

#pragma region //free cuda memory and printing execution time

    //hipFree(); 

    end=clock();
    cout << "Exe time: "<<(((double)(end-start))/CLOCKS_PER_SEC)<<" sec"<<endl;
#pragma region

    return 0;
}